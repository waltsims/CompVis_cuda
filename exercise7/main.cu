#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Winter Semester 2015/2016, March 15 - April 15
// ###
// ###

#include "helper.h"
#include <iostream>
using namespace std;

const float pi = 3.141592653589793238462;

texture<float, 2, hipReadModeElementType> texRef; // def at file scope

__constant__ float c_kernel[41 * 41 * sizeof(float)];

// uncomment to use the camera
//#define CAMERA

void createKernel(float *kernel, float *kernel_n, float sigma, int w, int h) {

  int mean = w / 2;
  float sum = 0.0;

  for (int i = 0; i < w; i++) {
    for (int j = 0; j < h; j++) {

      kernel[j + i * h] =
          (1.0f / (2.0f * pi * sigma * sigma)) *
          exp(-1 * (((i - mean) * (i - mean) + (j - mean) * (j - mean)) /
                    (2 * sigma * sigma)));

      sum += kernel[j + i * h];
    }
  }

  // normilize the kernel sum
  float max = 0.0;
  for (int i = 0; i < w; i++) {
    for (int j = 0; j < h; j++) {

      kernel[j + i * h] /= sum;

      if (kernel[j + i * h] > max)
        max = kernel[j + i * h];
      //cout << kernel[j * i * h] << " ,";
    }
    //cout << endl;
  }
  
    for (int i = 0; i < w; i++) {
      for (int j = 0; j < h; j++) {

        kernel_n[j + i * h] = kernel[j + i * h] / max;
      }
    }
}

__global__ void globalConvolution(float *d_imgIn, float *d_kernel, float *d_imgOut,
                            int nc, int w, int h, int w_k, int h_k) {
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  size_t ind = x + (size_t)w * y;

  int mid = w_k / 2;

  if (x < w && y < h) {
    for (int c = 0; c < nc; c++) {
      d_imgOut[ind + w * h * c] = 0;
      for (int k = 0; k < w_k; k++) {
        for (int l = 0; l < h_k; l++) {
          int i_k = x - mid + k;
          int j_k = y - mid + l;

          if (i_k < 0)
            i_k = 0;
          if (i_k > w - 1)
            i_k = w - 1;
          if (j_k > h - 1)
            j_k = h - 1;
          if (j_k < 0)
            j_k = 0;

          d_imgOut[ind + w * h * c] +=
              d_kernel[l * w_k + k] * d_imgIn[j_k * w + i_k + w * h * c];
        }
      }
    }
  }
}

__global__ void computeM(float *d_gradStencilX, float *d_gradStencilY,
                         float *d_m1, float *d_m2, float *d_m3, int w, int h,
                         int nc) {
  // image coordinates
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  size_t ind = x + (size_t)w * y;
  if (x < w && y < h) {
    d_m1[ind] = 0;
    d_m2[ind] = 0;
    d_m3[ind] = 0;

    for (int i = 0; i < nc; i++) {
      d_m1[ind] +=
          d_gradStencilX[ind + w * h * i] * d_gradStencilX[ind + w * h * i];
      d_m2[ind] +=
          d_gradStencilX[ind + w * h * i] * d_gradStencilY[ind + w * h * i];
      d_m3[ind] +=
          d_gradStencilY[ind + w * h * i] * d_gradStencilY[ind + w * h * i];
    }
  }
}

__global__ void textureConvolution(float *d_imgIn, float *d_kernel, float *d_imgOut,
                            int nc, int w, int h, int w_k, int h_k, int r, int mean) {
  // image coordinates
  int x = threadIdx.x + blockDim.x * blockIdx.x;
  int y = threadIdx.y + blockDim.y * blockIdx.y;
  // block coords
  int xblock = threadIdx.x; // local version of x
  int yblock = threadIdx.y; // local version if y

  // shared memore window dimensions
  int sw = blockDim.x + 2 * r; // calculate size of window
  int sh = blockDim.y + 2 * r;

  for (int c = 0; c < nc; c++) {

    if (x < w && y < h) {

      // each thread reads elements to sh_imgIn
      size_t ind = x + (size_t)w * y + w * h * c;
      d_imgOut[ind] = 0; // initialize the output data to zero

	  for (int k = 0; k < w_k; k++) {
		for (int l = 0; l < h_k; l++) {

		  int i_k = x - mean + k ;  //threadIdx.x + k;
		  int j_k = y - mean + l;  //threadIdx.y + l;

		  float val = tex2D(texRef, i_k + 0.5f, j_k + 0.5f +  h * c);

		  d_imgOut[ind] +=
			  c_kernel[l * w_k + k] * val; 
		}
	  }
    }
    __syncthreads();
  }
}

int main(int argc, char **argv) {
  // Before the GPU can process your kernels, a so called "CUDA context" must be
  // initialized
  // This happens on the very first call to a CUDA function, and takes some time
  // (around half a second)
  // We will do it right here, so that the run time measurements are accurate
  hipDeviceSynchronize();
  CUDA_CHECK;

// Reading command line parameters:
// getParam("param", var, argc, argv) looks whether "-param xyz" is specified,
// and if so stores the value "xyz" in "var"
// If "-param" is not specified, the value of "var" remains unchanged
//
// return value: getParam("param", ...) returns true if "-param" is specified,
// and false otherwise

#ifdef CAMERA
#else
  // input image
  string image = "";
  bool ret = getParam("i", image, argc, argv);
  if (!ret)
    cerr << "ERROR: no image specified" << endl;
  if (argc <= 1) {
    cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]"
         << endl;
    return 1;
  }
#endif

  // number of computation repetitions to get a better run time measurement
  int repeats = 1;
  getParam("repeats", repeats, argc, argv);
  cout << "repeats: " << repeats << endl;

  // load the input image as grayscale if "-gray" is specifed
  bool gray = false;
  getParam("gray", gray, argc, argv);
  cout << "gray: " << gray << endl;
  float sigma = 1.0;
  getParam("sigma", sigma, argc, argv);
  cout << "sigma: " << sigma << endl;
// ### Define your own parameters here as needed

// Init camera / Load input image
#ifdef CAMERA

  // Init camera
  cv::VideoCapture camera(0);
  if (!camera.isOpened()) {
    cerr << "ERROR: Could not open camera" << endl;
    return 1;
  }
  int camW = 640;
  int camH = 480;
  camera.set(CV_CAP_PROP_FRAME_WIDTH, camW);
  camera.set(CV_CAP_PROP_FRAME_HEIGHT, camH);
  // read in first frame to get the dimensions
  cv::Mat mIn;
  camera >> mIn;

#else

  // Load the input image using opencv (load as grayscale if "gray==true",
  // otherwise as is (may be color or grayscale))
  cv::Mat mIn =
      cv::imread(image.c_str(), (gray ? CV_LOAD_IMAGE_GRAYSCALE : -1));
  // check
  if (mIn.data == NULL) {
    cerr << "ERROR: Could not load image " << image << endl;
    return 1;
  }

#endif

  // convert to float representation (opencv loads image values as single bytes
  // by default)
  mIn.convertTo(mIn, CV_32F);
  // convert range of each channel to [0,1] (opencv default is [0,255])
  mIn /= 255.f;
  // get image dimensions
  int w = mIn.cols;        // width
  int h = mIn.rows;        // height
  int nc = mIn.channels(); // number of channels
  int r = ceil(3 * sigma);
  int w_k = r * 2 + 1;
  int h_k = w_k;
  cout << "image: " << w << " x " << h << endl;

  // Set the output image format
  // ###
  // ###
  // ### TODO: Change the output image format as needed
  // ###
  // ###
  cv::Mat mOut(h, w, mIn.type()); // mOut will have the same number of channels
                                  // as the input image, nc layers
  // define kernel image
  // cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
  // cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
  // ### Define your own output images here as needed
  cv::Mat mKern(h_k, w_k, CV_32FC1); // mOut will be a grayscale image, 1 layer
  cv::Mat mM1(h, w, CV_32FC1); // mOut will be a grayscale image, 1 layer
  cv::Mat mM2(h, w, CV_32FC1); // mOut will be a grayscale image, 1 layer
  cv::Mat mM3(h, w, CV_32FC1); // mOut will be a grayscale image, 1 layer

  // Allocate arrays
  // input/output image width: w
  // input/output image height: h
  // input image number of channels: nc
  // output image number of channels: mOut.channels(), as defined above (nc, 3,
  // or 1)

  // allocate raw input image array
  float *imgIn = new float[(size_t)w * h * nc];

  // allocate raw output array (the computation result will be stored in this
  // array, then later converted to mOut for displaying)
  float *imgOut = new float[(size_t)w * h * mOut.channels()];

// For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
  // Read a camera image frame every 30 milliseconds:
  // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
  // returns a value <0 if no key is pressed during this time, returns
  // immediately with a value >=0 if a key is pressed
  while (cv::waitKey(30) < 0) {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single
    // bytes by default)
    mIn.convertTo(mIn, CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg...
    // bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for
    // loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered(imgIn, mIn);

    Timer timer;
    timer.start();
    // ###
    // ###
    float *kernel = new float[w_k * w_k]; // height is same as width
    float *kernel_n = new float[w_k * w_k]; // height is same as width
	float *m1 = new float [w * h];
	float *m2 = new float [w * h];
	float *m3 = new float [w * h];

	float stencilX[9] = {-3 , 0, 3, -10, 0, 10, -3, 0, 3};
	float *gradStencilX = stencilX;
	float stencilY[9] = {-3, -10, -3, 0, 0, 0, 3, 10, 3};
	float *gradStencilY = stencilY;
	// normalize values
	for (int i = 0 ; i < 9 ; i ++){
		//cout<< "X_0: " << stencilX[i] << "; ";
		stencilX[i]	/= 32.0;
		stencilY[i]	/= 32.0;
		//cout<< "X_1: " << stencilX[i] << endl;
	}

	float *d_gradStencilX = new float[3 * 3];
	float *d_gradStencilY = new float[3 * 3];
	float *d_gradX;
	float *d_gradY;
	float *d_m1;
	float *d_m2;
	float *d_m3;
	float *d_m1c;
	float *d_m2c;
	float *d_m3c;
    float *d_kernel;
    float *d_imgIn;
    float *d_imgOut;


    createKernel(kernel, kernel_n, sigma, w_k, h_k);

    hipMalloc(&d_kernel, w_k * h_k * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_imgIn, nc * w * h * sizeof(float)); CUDA_CHECK;
    hipMalloc(&d_imgOut, nc * w * h * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_gradStencilX, 3 * 3 * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_gradStencilY, 3 * 3 * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_gradX, nc * w * h * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_gradY, nc * w * h * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_m1, w * h * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_m2, w * h * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_m3, w * h * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_m1c, w * h * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_m2c, w * h * sizeof(float)); CUDA_CHECK;
	hipMalloc(&d_m3c, w * h * sizeof(float)); CUDA_CHECK;


    hipMemcpy(d_kernel, kernel, w_k * h_k * sizeof(float),
               hipMemcpyHostToDevice); CUDA_CHECK; 
    hipMemcpy(d_imgIn, imgIn, nc * w * h * sizeof(float),
               hipMemcpyHostToDevice); CUDA_CHECK;

	hipMemcpy(d_gradStencilX, gradStencilX, 3 * 3 * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;
	hipMemcpy(d_gradStencilY, gradStencilY, 3 * 3 * sizeof(float), hipMemcpyHostToDevice); CUDA_CHECK;

    dim3 block = dim3(32, 8, 1); // 32*8 = 256 threads
    dim3 grid =
        dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, 1);
    size_t smBytes = (block.x + 2 * r) * (block.y + 2 * r) * sizeof(float);

    texRef.addressMode[0] = hipAddressModeClamp; // clamp x to border
    texRef.addressMode[1] = hipAddressModeClamp; // clampm y to border
    texRef.filterMode = hipFilterModeLinear;    // linear intermpolation
    // access as (x + 0.5f, y + 0.5f), not as ((x+0.5f)/w,(y+0.5f)/h
    texRef.normalized = false;

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	CUDA_CHECK;

    hipBindTexture2D(NULL, &texRef, d_imgIn, &desc, w, nc * h,
                      w * sizeof(d_imgIn[0]));
	CUDA_CHECK;
	
	hipMemcpyToSymbol(HIP_SYMBOL(c_kernel), kernel, w_k * h_k * sizeof(float));

    // calculate convolution!

    textureConvolution<<<grid, block, smBytes>>>(d_imgIn, d_kernel, d_imgOut, nc, w, h,
                                          w_k, h_k, r, w_k / 2);
    hipUnbindTexture(texRef);

    // calculate gradient of convoluted image
    // for X
    globalConvolution<<<grid, block >>>(d_imgOut, d_gradStencilX, d_gradX, nc, w,
                                       h, 3, 3);
    // for Y
    globalConvolution<<<grid, block>>>(d_imgOut, d_gradStencilY, d_gradY, nc, w,
                                       h, 3, 3);

    computeM<<<grid, block >>>(d_gradX, d_gradY, d_m1,
                                      d_m2, d_m3, w, h, nc);

	globalConvolution<<<grid, block>>>(d_m1, d_kernel, d_m1c, 1, w, h,
										  w_k, h_k);

	globalConvolution<<<grid, block>>>(d_m2, d_kernel, d_m2c, 1, w, h,
                                          w_k, h_k);

	globalConvolution<<<grid, block>>>(d_m3, d_kernel, d_m3c, 1, w, h,
                                          w_k, h_k);

    /*hipMemcpy(imgOut, d_imgOut, nc * w * h * sizeof(float),*/
               /*hipMemcpyDeviceToHost);*/
    hipMemcpy(imgOut, d_gradY , nc * w * h * sizeof(float),
               hipMemcpyDeviceToHost);
	hipMemcpy(m1, d_m1c ,w * h * sizeof(float),
			   hipMemcpyDeviceToHost);
	hipMemcpy(m2, d_m2c ,w * h * sizeof(float),
			   hipMemcpyDeviceToHost);
	hipMemcpy(m3, d_m3c ,w * h * sizeof(float),
			   hipMemcpyDeviceToHost);

    hipFree(d_imgIn);
    hipFree(d_imgOut);
    hipFree(d_kernel);
	//TODO free memory for stuff

    timer.end();
    float t = timer.get(); // elapsed time in seconds
    cout << "time: " << t * 1000 << " ms" << endl;

    // show input image
    showImage("Input", mIn, 100,
              100); // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the
    // layered raw array
    convert_layered_to_mat(mOut, imgOut);
    convert_layered_to_mat(mKern, kernel_n);
	convert_layered_to_mat(mM1, m1);
	convert_layered_to_mat(mM2, m2);
	convert_layered_to_mat(mM3, m3);
	
	//showImage("Output", mOut * 10 , 100 + w + 40, 100);
	showImage("m1", mM1 * 10 , 50, 100);
	showImage("m2", mM2 * 10 , 50 + w , 100);
	showImage("m3", mM3 * 10 , 50 + 2 * w , 100);
    //showImage("Gaussian Kernel", mKern, 100 + w + 40, 100);

// ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
  }
#else
  // wait for key inputs
  cv::waitKey(0);
#endif

  // save input and result
  cv::imwrite("image_input.png",
              mIn * 255.f); // "imwrite" assumes channel range [0,255]
  cv::imwrite("image_result.png", mOut * 255.f);

  // free allocated arrays
  delete[] imgIn;
  delete[] imgOut;

  // close all opencv windows
  cvDestroyAllWindows();
  return 0;
}

